#include "hip/hip_runtime.h"
#include "keypoint_detector.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <stdio.h>

#define MAX_DETECTORS 16



// ============================================================= Detection =================================================================

// FAST circle offsets
__constant__ int2 fast_offsets[16] = {
    {3,  0},  {3,  1},  {2,  2},  {1,  3},
    {0,  3},  {-1, 3},  {-2, 2},  {-3, 1},
    {-3, 0},  {-3, -1}, {-2, -2}, {-1, -3},
    {0, -3},  {1, -3},  {2, -2},  {3,  -1}
};


// Global state
static DetectorInstance g_detectors[MAX_DETECTORS] = {0};
static int g_next_detector_id = 0;

static DetectorInstance* get_detector_instance(int id) {
    for (int i = 0; i < MAX_DETECTORS; i++) {
        if (g_detectors[i].initialized && g_detectors[i].id == id) {
            return &g_detectors[i];
        }
    }
    return NULL;
}

static int find_free_detector_slot(void) {
    for (int i = 0; i < MAX_DETECTORS; i++) {
        if (!g_detectors[i].initialized) {
            return i;
        }
    }
    return -1;
}

__device__ float3 convertImageToWorldCoords(float x, float y, float imageWidth, float imageHeight) {
    float normalizedX = (x / imageWidth) * 2.0f - 1.0f;
    float normalizedY = -((y / imageHeight) * 2.0f - 1.0f);
    
    float worldX = normalizedX * 6.4f;
    float worldY = normalizedY * 3.6f;
    
    return make_float3(worldX, -0.01f, worldY);
}

__global__ void detectFASTKeypoints(
    const uint8_t* __restrict__ y_plane,
    int width,
    int height,
    int linesize,
    uint8_t threshold,
    float4* positions,
    float4* colors,
    int* keypoint_count,
    int max_keypoints,
    float image_width,
    float image_height
) {
    __shared__ int block_counter;
    __shared__ float2 block_keypoints[256]; // Adjust size based on block size

    if (threadIdx.x == 0) {
        block_counter = 0;
    }
    __syncthreads();

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 3 || y < 3 || x >= width - 3 || y >= height - 3) return;

    const uint8_t center = y_plane[y * linesize + x];

    int brighter = 0;
    int darker = 0;

    for (int i = 0; i < 16; i++) {
        const int2 offset = fast_offsets[i];
        const uint8_t pixel = y_plane[(y + offset.y) * linesize + (x + offset.x)];

        if (pixel > center + threshold) brighter++;
        else if (pixel < center - threshold) darker++;
    }

    bool is_keypoint = (brighter >= 9 || darker >= 9);

    if (is_keypoint) {
        int local_idx = atomicAdd(&block_counter, 1);
        if (local_idx < 256) { // Ensure local storage doesn't overflow
            block_keypoints[local_idx] = make_float2(x, y);
        }
    }
    __syncthreads();

   if (threadIdx.x == 0 && block_counter > 0) {
        int global_idx = atomicAdd(keypoint_count, block_counter);
        if (global_idx + block_counter <= max_keypoints) {
            for (int i = 0; i < block_counter; i++) {
                float2 kp = block_keypoints[i];
                float3 world_pos = convertImageToWorldCoords(
                    kp.x, kp.y, 
                    image_width, image_height
                );
                positions[global_idx + i] = make_float4(world_pos.x, world_pos.y, world_pos.z, 1.0f);
                colors[global_idx + i] = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
            }
        }
    }
}


// ============================================================= Matching =================================================================

// Matching parameters
struct MatchingParams {
    float baseline;
    float focal_length;
    float max_disparity;
    float epipolar_threshold;
    float sensor_width_mm;
    float sensor_width_pixels;
    float sensor_height_pixels;
};

// Structure to hold matched keypoint data
struct MatchedKeypoint {
    float3 left_pos;
    float3 right_pos;
    float3 world_pos;
    float disparity;
};

struct MatchedPoint {
    float3 position;    // Position in OpenGL world coordinates
    float disparity;    // Pixel disparity between left and right views
};

__device__ MatchedPoint calculateMatchedWorldPosition(
    float3 leftWorldPos,   // Already transformed by convertImageToWorldCoords
    float3 rightWorldPos,  // Already transformed by convertImageToWorldCoords
    float baseline,        // Distance between cameras in world units
    float canvas_width     // Width of the canvas in world units
) {
    // Calculate disparity in world units
    float worldDisparity = leftWorldPos.x - rightWorldPos.x;
    
    // Since the positions are already in world coordinates, we can use them directly
    // But we need to account for the baseline shift and calculate the midpoint
    MatchedPoint result;
    
    // Calculate the matched point position as the midpoint between left and right points,
    // but adjusted for depth based on disparity
    float depthFactor = baseline / worldDisparity;
    
    // X position: average of left and right X coordinates
    float worldX = (leftWorldPos.x + rightWorldPos.x) / 2.0f;
    
    // Y position: keep consistent with your original function
    float worldY = -0.01f;
    
    // Z position: average of left and right Z coordinates, scaled by depth factor
    float worldZ = (leftWorldPos.z + rightWorldPos.z) / 2.0f;
    worldZ *= depthFactor;
    
    // Ensure the position stays within the canvas bounds
    worldX = fmax(fmin(worldX, canvas_width/2), -canvas_width/2);
    worldZ = fmax(fmin(worldZ, canvas_width/2), -canvas_width/2);
    
    result.position = make_float3(worldX, worldY, worldZ);
    result.disparity = worldDisparity;
    
    return result;
}

__global__ void matchKeypointsKernel(
    const float4* __restrict__ left_positions,
    const float4* __restrict__ right_positions,
    const int left_count,
    const int right_count,
    const MatchingParams params,
    MatchedKeypoint* matches,
    int* match_count,
    const int max_matches
) {
    const int left_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (left_idx >= left_count) return;

    __shared__ float min_costs[512]; 
    __shared__ int best_matches[512];

    min_costs[threadIdx.x] = INFINITY;
    best_matches[threadIdx.x] = -1;

    float3 left_pos = make_float3(
        left_positions[left_idx].x,
        left_positions[left_idx].y,
        left_positions[left_idx].z
    );

    // Find best match for this left keypoint
    for (int right_idx = 0; right_idx < right_count; right_idx++) {
        float3 right_pos = make_float3(
            right_positions[right_idx].x,
            right_positions[right_idx].y,
            right_positions[right_idx].z
        );

        // Check epipolar constraint
        float y_diff = fabsf(left_pos.y - right_pos.y);
        if (y_diff > params.epipolar_threshold) continue;

        // Calculate disparity (should be positive)
        float disparity = left_pos.x - right_pos.x;
        if (disparity <= 0 || disparity > params.max_disparity) continue;

        // Simple cost function based on y-difference
        float cost = y_diff;

        if (cost < min_costs[threadIdx.x]) {
            min_costs[threadIdx.x] = cost;
            best_matches[threadIdx.x] = right_idx;
        }
    }

    __syncthreads();

    // Store match if good enough
    if (best_matches[threadIdx.x] >= 0 && min_costs[threadIdx.x] < params.epipolar_threshold) {
        int match_idx = atomicAdd(match_count, 1);
        if (match_idx < max_matches) {
           float3 right_pos = make_float3(
                right_positions[best_matches[threadIdx.x]].x,
                right_positions[best_matches[threadIdx.x]].y,
                right_positions[best_matches[threadIdx.x]].z
            );

            MatchedPoint matchedPoint = calculateMatchedWorldPosition(
                left_pos,
                right_pos,
                params.baseline,  // This should be baseline converted to world units
                6.4f                    // Canvas width in world units
            );

            matches[match_idx] = {
                left_pos,
                right_pos,
                matchedPoint.position,
                matchedPoint.disparity
            };
        }
    }
}

// Kernel to generate visualization data
__global__ void generateVisualizationKernel(
    const MatchedKeypoint* matches,
    const int match_count,
    float4* positions,
    float4* colors
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= match_count) return;

    const MatchedKeypoint match = matches[idx];
    const int base_idx = idx; // 3 vertices per match (triangle strip)

    // Center point (world position)
    positions[base_idx] = make_float4(match.world_pos.x, match.world_pos.y, match.world_pos.z, 1.0f);
    colors[base_idx] = make_float4(0.0f, 1.0f, 0.0f, 1.0f); // Green for center

    // Left keypoint
    // positions[base_idx + 1] = make_float4(match.left_pos.x, match.left_pos.y, match.left_pos.z, 1.0f);
    // colors[base_idx + 1] = make_float4(1.0f, 0.0f, 0.0f, 1.0f); // Red for left

    // // Right keypoint
    // positions[base_idx + 2] = make_float4(match.right_pos.x, match.right_pos.y, match.right_pos.z, 1.0f);
    // colors[base_idx + 2] = make_float4(0.0f, 0.0f, 1.0f, 1.0f); // Blue for right
}



// ============================================================= Bindings =================================================================

extern "C" {

int cuda_create_detector(void) {
    int slot = find_free_detector_slot();
    if (slot < 0) {
        return -1;
    }

    if (hipMalloc(&g_detectors[slot].d_keypoint_count, sizeof(int)) != hipSuccess) {
        return -1;
    }

    g_detectors[slot].initialized = true;
    g_detectors[slot].id = g_next_detector_id++;
    return g_detectors[slot].id;
}

int cuda_register_gl_buffers(int detector_id, unsigned int position_buffer, unsigned int color_buffer, int max_keypoints) {
    DetectorInstance* detector = get_detector_instance(detector_id);
    if (!detector) return -1;

    hipError_t error;

    // Check which CUDA device is currently in use
    int cudaDevice;
    error = hipGetDevice(&cudaDevice);
    if (error != hipSuccess) {
        printf("Error getting CUDA device: %s\n", hipGetErrorString(error));
        return -1;
    }

    hipDeviceProp_t prop;
    error = hipGetDeviceProperties(&prop, cudaDevice);
    if (error != hipSuccess) {
        printf("Error getting CUDA device properties: %s\n", hipGetErrorString(error));
        return -1;
    }

    printf("Using CUDA Device: %d - %s\n", cudaDevice, prop.name);

    // Register position buffer
    error = hipGraphicsGLRegisterBuffer(
        &detector->gl_resources.position_resource,
        position_buffer,
        hipGraphicsRegisterFlagsWriteDiscard
    );
    if (error != hipSuccess) {
        printf("Error registering Position Buffer %d => %d\n", position_buffer, error);
        return -1;
    }

    // Register color buffer
    error = hipGraphicsGLRegisterBuffer(
        &detector->gl_resources.color_resource,
        color_buffer,
        hipGraphicsRegisterFlagsWriteDiscard
    );
    if (error != hipSuccess) {
        hipGraphicsUnregisterResource(detector->gl_resources.position_resource);
        printf("Error registering Color Buffer %d => %d\n", color_buffer, error);
        return -1;
    }

    detector->gl_resources.buffer_size = max_keypoints;
    return 0;
}

int cuda_detect_keypoints(
    int detector_id,
    uint8_t threshold,
    ImageParams* image
) {
    DetectorInstance* detector = get_detector_instance(detector_id);
    
    if (!detector) return -1;

    hipError_t error;
 
    dim3 block(16, 16);
    dim3 grid((image->width + block.x - 1) / block.x, (image->height + block.y - 1) / block.y);

    // Reset keypoint counter
    error = hipMemset(detector->d_keypoint_count, 0, sizeof(int));
    if (error != hipSuccess) {
        printf("Failed to reset keypoint count: %d\n", error);
        return -1;
    }

    // Launch kernel
    detectFASTKeypoints<<<grid, block>>>(
        image->y_plane,
        image->width,
        image->height,
        image->y_linesize,
        threshold,
        detector->gl_resources.d_positions,
        detector->gl_resources.d_colors,
        detector->d_keypoint_count,
        detector->gl_resources.buffer_size,
        image->image_width,
        image->image_height
    );


    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Keypoint Detection Kernel failed: %d\n", error);
        return -1;
    }


    // Get keypoint count
    error = hipMemcpy(image->num_keypoints, detector->d_keypoint_count, sizeof(int), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        printf("Failed to copy keypoint count: %d\n", error);
        return -1;
    }
    
    return error == hipSuccess ? 0 : -1;
}


void cuda_unregister_gl_buffers(int detector_id) {
    DetectorInstance* detector = get_detector_instance(detector_id);
    if (!detector) return;

    if (detector->gl_resources.position_resource) {
        hipGraphicsUnregisterResource(detector->gl_resources.position_resource);
    }
    if (detector->gl_resources.color_resource) {
        hipGraphicsUnregisterResource(detector->gl_resources.color_resource);
    }
    detector->gl_resources = {};
}

void cuda_cleanup_detector(int detector_id) {
    DetectorInstance* detector = get_detector_instance(detector_id);
    if (!detector) return;

    cuda_unregister_gl_buffers(detector_id);
    if (detector->d_keypoint_count) hipFree(detector->d_keypoint_count);
    detector->d_keypoint_count = nullptr;
}


int cuda_map_gl_resources(int detector_id) {
    DetectorInstance* detector = get_detector_instance(detector_id);
    if (!detector) return -1;

    // Map GL buffers for CUDA access
    hipError_t error = hipGraphicsMapResources(1, &detector->gl_resources.position_resource);
    if (error != hipSuccess) {
        printf("Failed to Map Position Resources: %d\n", error);
        return -1;
    };

    error = hipGraphicsMapResources(1, &detector->gl_resources.color_resource);
    if (error != hipSuccess) {
        printf("Failed to Map Color Resources: %d\n", error);
        hipGraphicsUnmapResources(1, &detector->gl_resources.position_resource);
        return -1;
    }
        
    size_t bytes;
    error = hipGraphicsResourceGetMappedPointer(
        (void**)&detector->gl_resources.d_positions,
        &bytes,
        detector->gl_resources.position_resource
    );

    if (error != hipSuccess) {
        printf("Failed to get Positions Mapped Pointer: %d\n", error);
        cuda_unmap_gl_resources(detector_id);
    }

    error = hipGraphicsResourceGetMappedPointer(
        (void**)&detector->gl_resources.d_colors,
        &bytes,
        detector->gl_resources.color_resource
    );

    if (error != hipSuccess) {
        printf("Failed to get Colors Mapped Pointer: %d\n", error);
        cuda_unmap_gl_resources(detector_id);
    }

    return error == hipSuccess ? 0 : -1;
}

void cuda_unmap_gl_resources(int detector_id) {
    DetectorInstance* detector = get_detector_instance(detector_id);
    if (!detector) return;

    hipGraphicsUnmapResources(1, &detector->gl_resources.position_resource);
    hipGraphicsUnmapResources(1, &detector->gl_resources.color_resource);
}

int cuda_match_keypoints(
    int detector_id_left,
    int detector_id_right,
    int detector_id_combined,
    float baseline,
    float focal_length,
    int* num_matches,
    uint8_t threshold,

    ImageParams* left,
    ImageParams* right
) {
    DetectorInstance* left_detector = get_detector_instance(detector_id_left);
    DetectorInstance* right_detector = get_detector_instance(detector_id_right);
    DetectorInstance* combined_detector = get_detector_instance(detector_id_combined);

    if (!left_detector || !right_detector || !combined_detector) return -1;
    
    
    if (cuda_map_gl_resources(detector_id_left) < 0){
        printf("Failed to map GL Resources for Left Detector!\n");
        return -1;
    }

    printf("Getting keypoints from left...\n");
    int result = cuda_detect_keypoints(
        detector_id_left,
        threshold,
        left
    );

    if (result < 0){
        printf("Failed to detect keypoints from left image\n");
        cuda_unmap_gl_resources(detector_id_left);
        return -1;
    };

    if (cuda_map_gl_resources(detector_id_right) < 0){
        printf("Failed to map GL Resources for Right Detector!\n");
        cuda_unmap_gl_resources(detector_id_left);
        return -1;
    }

    printf("Getting keypoints from right...\n");
    result = cuda_detect_keypoints(
        detector_id_right,
        threshold,
        right
    );

    if (result < 0){
        printf("Failed to detect keypoints from right image\n");
        cuda_unmap_gl_resources(detector_id_left);
        cuda_unmap_gl_resources(detector_id_right);
        return -1;
    };

    hipDeviceSynchronize();
    
    //Use left_detector as the basis for matching
    const int max_matches = min(*right->num_keypoints, *left->num_keypoints);

    printf("Left keypoints: %d\n", *left->num_keypoints);
    printf("Right Keypoints: %d\n", *right->num_keypoints);
    printf("Max matches allowed: %d\n", max_matches);
   
    // Allocate device memory for matches
    MatchedKeypoint* d_matches;
    int* d_match_count;
    
    hipMalloc(&d_matches, max_matches * sizeof(MatchedKeypoint));
    hipMalloc(&d_match_count, sizeof(int));
    hipMemset(d_match_count, 0, sizeof(int));

    // Set up matching parameters
    float sensor_width_mm = 6.4f;
    float baseline_world = (76.3f / sensor_width_mm) * 6.4f;

    MatchingParams params = {
        .baseline = baseline_world,  // mm
        .focal_length = 3.2f,        // mm
        .max_disparity = 100.0f,     // pixels
        .epipolar_threshold = 2.0f,  // pixels
        .sensor_width_mm = sensor_width_mm,     // mm
        .sensor_width_pixels = 4608.0f,  // pixels
        .sensor_height_pixels = 2592.0f  // pixels
    };

    // Launch matching kernel
    dim3 blockA(512);
    dim3 gridA((max_matches + blockA.x - 1) / blockA.x); 

    printf("Matching Dims: Thread per Block: %d : Blocks: %d => Total Threads: %d\n", blockA.x, gridA.x, blockA.x * gridA.x);

    matchKeypointsKernel<<<blockA, gridA>>>(
        left_detector->gl_resources.d_positions,
        right_detector->gl_resources.d_positions,
        *left->num_keypoints,
        *right->num_keypoints,
        params,
        d_matches,
        d_match_count,
        max_matches
    );

    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Matching Kernel failed: %s\n", hipGetErrorString(error));
        return -1;
    }

    // Get match count
    hipMemcpy(num_matches, d_match_count, sizeof(int), hipMemcpyDeviceToHost);
    printf("Detected Matches: %d\n", *num_matches);
    
    if (cuda_map_gl_resources(detector_id_combined) < 0){
        printf("Failed to map GL Resources for Combined Detector!\n");
        cuda_unmap_gl_resources(detector_id_left);
        cuda_unmap_gl_resources(detector_id_right);
        return -1;
    }
    
    dim3 blockB(1024);
    dim3 gridB(((*num_matches) + blockB.x - 1) / blockB.x); 
    
    // Generate visualization
    generateVisualizationKernel<<<blockB, gridB>>>(
        d_matches,
        *num_matches,
        combined_detector->gl_resources.d_positions,
        combined_detector->gl_resources.d_colors
    );

    hipDeviceSynchronize();

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Visualization Kernel failed: %s\n", hipGetErrorString(error));
    }


    hipFree(d_matches);
    hipFree(d_match_count);

    
    cuda_unmap_gl_resources(detector_id_left);
    cuda_unmap_gl_resources(detector_id_right);
    cuda_unmap_gl_resources(detector_id_combined);

    return error == hipSuccess ? 0 : -1;
}

}